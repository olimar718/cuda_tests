// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

const int N = 16; 
const int blocksize = 16;

__global__ 
void simple(float *cd) 
{
	cd[threadIdx.x] = pow(cd[threadIdx.x],2);
	printf("bonjour\n");
	cd[threadIdx.x] = sqrt(cd[threadIdx.x]);
}

int main()
{
	float *c = new float[N];
	float *cd;
	const int size = N*sizeof(float);

	for(int i=0; i<N; i++){
		c[i] = i*2;
	}	
	
	hipMalloc( (void**)&cd, size );
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );

	hipMemcpy(cd, c, size, hipMemcpyHostToDevice);

	simple<<<dimGrid, dimBlock>>>(cd);
	hipDeviceSynchronize();
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
	hipFree( cd );
	
	for (int i = 0; i < N; i++){
		printf("%f ", c[i]);
	}
	printf("\n");
	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
