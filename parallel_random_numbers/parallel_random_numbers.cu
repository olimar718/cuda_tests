//https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>     /* srand, rand */
#define N_THREADS (1<<10)
#define SIZE (int)(N_THREADS*sizeof(int))
#define N_COLOR 4

__global__ 
void random(int *seeds_gpu,int *randoms_gpu) 
{
	int result = seeds_gpu[threadIdx.x];
    result ^= result << 13;
    result ^= result >> 17;
    result ^= result << 5;
	randoms_gpu[threadIdx.x]=(result % N_COLOR+N_COLOR)%N_COLOR; //https://stackoverflow.com/questions/14997165/fastest-way-to-get-a-positive-modulo-in-c-c

}

int main()
{	
	printf("Number of threads %i\n",N_THREADS);
	int seeds[N_THREADS];
	int randoms[N_THREADS];

	for(int i = 0; i<N_THREADS;++i){
		seeds[i]=rand();
	}
	
	int *seeds_gpu;
	int *randoms_gpu;
	hipMalloc( (void**)&seeds_gpu, SIZE );
	hipMalloc( (void**)&randoms_gpu, SIZE );
	hipMemcpy(seeds_gpu, seeds, SIZE, hipMemcpyHostToDevice);


	dim3 dimBlock( N_THREADS ,1 , 1);
	dim3 dimGrid( 1, 1, 1 );
	random<<<dimGrid, dimBlock>>>(seeds_gpu,randoms_gpu);
	hipDeviceSynchronize();

	hipMemcpy( randoms, randoms_gpu, SIZE, hipMemcpyDeviceToHost ); 

	hipFree( randoms_gpu );
	hipFree( seeds_gpu );

	for (int i = 0; i < N_THREADS; i++){
		printf("generated random number : %i\n",randoms[i]);
	}
	return EXIT_SUCCESS;
}
