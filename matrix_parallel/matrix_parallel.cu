// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

const int N = 16; 
const int blocksize = 16;

__global__
void add_matrix(float *a, float *b, float *c)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.y * N;
    c[idx] = a[idx] + b[idx];
}

int main()
{
	const int N = 16;
    const int size = N*sizeof(float)*N;
    float *a_gpu;
    float *b_gpu;
    float *c_gpu;

	float *a = new float[N*N];
	float *b= new float[N*N];
    float *c= new float[N*N];

    for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = float(j) / N;
		}
	}
	hipMalloc( (void**)&a_gpu, size );
    hipMalloc( (void**)&b_gpu, size );
    hipMalloc( (void**)&c_gpu, size );
	dim3 dimBlock( blocksize, blocksize );
	dim3 dimGrid( 1, 1 );

	hipMemcpy(a_gpu, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, size, hipMemcpyHostToDevice);
    hipMemcpy(c_gpu, c, size, hipMemcpyHostToDevice);

    add_matrix<<<dimBlock, dimGrid>>>(a_gpu, b_gpu, c_gpu);
	hipDeviceSynchronize();
	hipMemcpy( c, c_gpu, size, hipMemcpyDeviceToHost ); 
	hipFree( a_gpu );
    hipFree( b_gpu );
    hipFree( c_gpu );

	
	for (int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
		    printf("%f ", c[i+j*N]);
        }
        printf("\n");
	}
	printf("\n");
	delete[] c;
    delete[] a;
    delete[] b;

	printf("done\n");
	return EXIT_SUCCESS;
}
