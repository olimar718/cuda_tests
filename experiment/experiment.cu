// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>     /* srand, rand */
//https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf
__global__ 
void experiment() 
{
	printf("blockIdx.x %i\n",blockIdx.x);
 	printf("blockIdx.y %i\n",blockIdx.y);
	printf("blockIdx.z %i\n",blockIdx.z);
	printf("\n");

	printf("blockDim.x %i\n",blockDim.x);
	printf("blockDim.y %i\n",blockDim.y);
	printf("blockDim.z %i\n",blockDim.z);
	printf("\n");

	printf("threadIdx.x %i\n",threadIdx.x);
	printf("threadIdx.y %i\n",threadIdx.y);
	printf("threadIdx.z %i\n",threadIdx.z);
	printf("\n");

	printf("gridDim.x %i\n",gridDim.x);
	printf("gridDim.y %i\n",gridDim.y);
	printf("gridDim.z %i\n",gridDim.z);
	printf("\n");



}

int main()
{	
	dim3 dimBlock( 1 ,1 ,2 );
	dim3 dimGrid( 1, 1, 1 );
	experiment<<<dimGrid, dimBlock>>>();
	hipDeviceSynchronize();
	return EXIT_SUCCESS;
}
