#include "hip/hip_runtime.h"
//https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>
#include <stdlib.h>     /* srand, rand */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define N_THREADS (1<<10)
#define SIZE (int)(N_THREADS*sizeof(int))
#define N_COLOR 4

void histogram( int* h_nums)
{
    int hist[N_COLOR + 1]; // from 0 to N_COLOR, so (N_COLOR + 1) elements needed
    for (int i = 0; i < N_COLOR + 1; i++)
        hist[i] = 0;
    for (int i = 0; i < N_THREADS; i++)
        hist[h_nums[i]]++;
    for (int i = 0; i < N_COLOR + 1; i++)
        printf("%2d : %6d\n", i, hist[i]);
}

__global__ void init(unsigned int seed, hiprandState_t* states)
{
    unsigned int id = threadIdx.x;
    hiprand_init(seed, /* the seed can be the same for each thread, here we pass the time from CPU */
                id,   /* the sequence number should be different for each core */
                0,    /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                &states[id]);
}

__global__ 
void random(int *randoms_gpu, hiprandState_t *  states) 
{
	unsigned int id=threadIdx.x;
	randoms_gpu[id]=hiprand_uniform(&states[id]) * (N_COLOR) ;
	//ternary set     b=( a == 5 ) ? c : b; to avoid branches
}

int main()
{	
	printf("Number of threads %i\n",N_THREADS);
	int randoms[N_THREADS];
	int seed=time(0);
	printf("seed : %i\n",seed);
	hiprandState_t* states;
    hipMalloc((void**)&states, N_THREADS * sizeof(hiprandState_t));
	int *randoms_gpu;
	hipMalloc( (void**)&randoms_gpu, SIZE );
	dim3 dimBlock( N_THREADS ,1 , 1);
	dim3 dimGrid( 1, 1, 1 );
    init<<<dimGrid, dimBlock >>>(time(0), states);
	random<<<dimGrid,dimBlock>>>(randoms_gpu,states);
	hipDeviceSynchronize();
	hipMemcpy( randoms, randoms_gpu, SIZE, hipMemcpyDeviceToHost ); 

	hipFree( randoms_gpu );
	hipFree( states );

	
	histogram(randoms);
	return EXIT_SUCCESS;
}
